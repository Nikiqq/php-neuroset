#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <sys/timeb.h>

void Multiplication(float *__restrict__ c, float * a, float * b, int N)
{
#pragma acc parallel loop present(c, a, b)
	for (int n = 0; n < N; n++)
	{
		for (int m = 0; m < N; m++)
		{
			float sum = 0.f;
			for (int k = 0; k < N; k++)
			{
				sum += a[k + n * N] * b[k * N + m ];
			}
			c[m + n * N] = sum;
		}
	}
}
int main()
{
	int i;
	const int N = 4;
	float **Matrix_A = (float**)malloc(N * sizeof(float*));
	for (i = 0; i < N; i++) {
		Matrix_A[i] = (float*)malloc(N * sizeof(float*));
	}
    
	float **Matrix_B = (float**)malloc(N * sizeof(float*));
	for (i = 0; i < N; i++) {
		Matrix_B[i] = (float*)malloc(N * sizeof(float*));
	}
    
	float **Matrix_C = (float**)malloc(N * sizeof(float*));
	for (i = 0; i < N; i++) {
		Matrix_C[i] = (float*)malloc(N * sizeof(float*));
	}
    
	float 
        * a = (float *)malloc(N * N * sizeof(float*)),
		* b = (float *)malloc(N * N * sizeof(float*)),
		* c = (float *)malloc(N * N * sizeof(float*));
        
    srand(time(NULL));
    
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < N; j++) {
			Matrix_A[i][j] = rand() % 5;
			Matrix_B[i][j] = rand() % 10;
		}
	}
    
	for (int i = 0; i < N; ++i) {
		for (int j = 0; j < N; j++) {
			a[j + i * N] = Matrix_A[i][j];
			b[j + i * N] = Matrix_B[i][j];
		}
	}
	
		
#pragma acc data copyin (a[0:N*N], b[0:N*N]) copyout (c[0:N*N])
	{
		Multiplication(c, a, b, N);
	}

    for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			printf("%3.0f  ", Matrix_A[i][j]);
		}
		printf("\n");
	}
    
    printf("\n");
    
    for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			printf("%3.0f  ", Matrix_B[i][j]);
		}
		printf("\n");
	}
    
    printf("\n");
    
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < N; j++) {
			Matrix_C[i][j] = c[j + i * N];
			printf("%3.0f  ", Matrix_C[i][j]);
		}
		printf("\n");
	}
	
	free(a);
	free(b);
	free(c);
	free(Matrix_A);
	free(Matrix_B);
	free(Matrix_C);
}