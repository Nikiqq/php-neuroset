#include "hip/hip_runtime.h"
#include <conio.h>
#include <iostream>
#include <string>
#include <cstdlib>
#include <ctime>
#include "hip/hip_runtime.h"
#include ""


__global__ void calculate(const char *input, char *output)
{
    char *s = '0';
    strcat(input, s);
    strcat(output, input);
}

using namespace std;

int main()
{
	setlocale(LC_ALL, "Russian");

	string prevBlock = "000000cdccf49f13f5c3f14a2c12a56ae60e900c5e65bfe1cc24f038f0668a6c";
	string timestamp = "01/01/2017";
	int nonce = 0;
	
	string init = prevBlock + timestamp;
	char * data_in = new char[init.length() + 1];
	strcpy(data_in, init.c_str());

	char * data_out = new char[init.length() + 2];

	char *data_in_gpu;
	char *data_out_gpu;

	hipMalloc((void **)&data_in_gpu, sizeof(char) * init.length() + 1);
	hipMalloc((void **)&data_out_gpu, sizeof(char) * init.length() + 2);

	hipMemcpy(data_in_gpu, data_in, sizeof(char) * init.length() + 1, hipMemcpyHostToDevice);

	dim3 gridDim(1);
	dim3 blockDim(1);

	calculate < << gridDim, blockDim >> >(data_in_gpu, data_out_gpu);

	hipMemcpy(data_out, data_out_gpu, sizeof(char) * init.length() + 2, hipMemcpyDeviceToHost);

	for (int i = 0; i < init.length() + 2; i++) {
		cout << data_out[i];
	}

	hipFree(data_in_gpu);
	hipFree(data_out_gpu);
	

	delete data_in;
	delete data_out;

	_getch(); // 
	return 0;
}