#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <random>
#include <bitset>
#include <iostream>
#include <ctime>

using namespace std;
const size_t length = 400;
const size_t bytel = length / 8;
using mes_type = std::bitset<length>;

__global__ void computeHash(unsigned char * arr, bool *  finds, int hence)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	finds[index] = false;
	unsigned char local_arr[bytel], second_arr[bytel];
	memcpy(local_arr, arr, length / 8);
	unsigned int nonce = hence + blockIdx.x * blockDim.x + threadIdx.x;
	//printf("nonce - %d\n", nonce);
	memcpy(local_arr + 1, &nonce, sizeof(unsigned int));
	int counter1 = 0;
	//printf("%d\n", sizeof(int));
	for (int i = 0; i < 5; ++i)
	{
		printf("%d\t", local_arr[i]);
	}
	printf("\n");
	short sum = 0;
	for (int j = 0; j < 3; ++j)
	{
		memcpy(second_arr, local_arr, bytel);
		for (int k = 0; k < 16; ++k)
		{
			unsigned char last_el = local_arr[bytel - 1] & 1, bits1 = 0, bits2 = 0;
			for (int i = 0; i < bytel; ++i)
			{
				bits2 = local_arr[i] & 1;
				local_arr[i] >>= 1;
				local_arr[i] |= bits1 << 7;
				bits1 = bits2;
			}
			local_arr[0] |= last_el << 7;
		}

		for (int f = bytel - 1; f >= 0; --f)
		{
		//	printf("%d + %d = ", second_arr[f], local_arr[f]);
			sum = second_arr[f] + local_arr[f] + sum;
			second_arr[f] = sum;
			sum >>= 8;
			sum &= 1;
		//	printf("%d\n", second_arr[f]);
		}
	}
	for (int i = 0; i < bytel; ++i)
	{
		printf("%i\t%d\n", i, second_arr[i]);
	}

	printf("%d\t%d\t", second_arr[bytel - 1], second_arr[0]);
	unsigned char buffer[8]{ second_arr[bytel - 1], second_arr[0]};
	int counter = 2, start = 8;
	for (int i = 0; i < 2; ++i)
	{
		buffer[2 + i * 3] = second_arr[1 + i];
		buffer[3 + i * 3] = second_arr[bytel / 2 + i];
		buffer[4 + i * 3] = second_arr[bytel - 3 + i];
		printf("%d\t%d\t%d\t", second_arr[1 + i], second_arr[bytel / 2 + i], second_arr[bytel - 3 + i]);
	}
	printf("\n");
	unsigned long long int number, remainder = 0;
	memcpy(&number, buffer, 8);

	remainder = number / 4294967296;
	if (remainder == 0) 
		finds[index] = true;
	printf("\n%llu -- %llu \n", number, remainder);
	//printf("%u\n",number);
}

int main()
{
	std::random_device r;
	std::mt19937 gen(r());
	std::bernoulli_distribution d(0.5);

	unsigned long long now = (unsigned long long)time(nullptr);
	const int len = sizeof(__int64) * 8;
	std::bitset<len> timestap( now );
	std::bitset<length> message;
	for (int i = 40; i < len + 40; ++i)
	{
		message[i] = (bool)timestap[i - 40];
	}
	for (int i = len; i < length; ++i)
	{
		message[i] = d( gen );
	}

	std::bitset<8> byte;
	unsigned char * arr = new unsigned char[length / 8], * cuda_message;
	bool some_var = true;
	cout << sizeof(int) << endl;
	for (int i = 0; i < length; i += 8)
	{
		for (int j = 0; j < 8; ++j)
		{
			some_var = (bool)message[j];
			byte[j] = some_var;
		}

		message >>= 8;
		arr[i / 8] = (char)byte.to_ulong();
		//cout << byte << " ";// << (int)(5 + arr[i / 8]) << endl;
		//printf("%u\n", arr[i / 8]);

	}
	unsigned long long int * cuda_key, host_key;

	int size = 256 * 512, hence = pow(16, 6);
	bool *  finds, * host_finds = new bool[size];
	hipMalloc((void**)&finds, size * sizeof(bool));
	hipMalloc((void**)&cuda_key, 8);
	hipMalloc((void**)&cuda_message, length / 8 + 1);
	hipMemcpy(cuda_message, arr, length / 8 + 1, hipMemcpyHostToDevice);

	for (int counter = 0; counter < 1; ++counter)
	{
		cout << "epoch " << counter << endl;
		computeHash<<<1, 1 >>>(cuda_message, finds, hence * (counter + 1));
		hipDeviceSynchronize();
		
		
		//cout << "--------" << endl;
		bool err = hipMemcpy(host_finds, finds, size * sizeof(bool), hipMemcpyDeviceToHost);
		for (int i = 0; i < size; ++i)
		{
			if (host_finds[i])
				cout << "thread " << i << " found it" << endl;
		}
	}
	cout << "ended" << endl;
	return 0;
}
