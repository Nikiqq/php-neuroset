#include "hip/hip_runtime.h"
#include <conio.h>
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <fstream>
#include <iomanip>
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <sys/timeb.h>

using namespace std;

const double step_t = 0.1;
const double step_xyz = 0.1;
const int Nt = 10;
const int rasmer = 3;
const int Nxyz = 10;
const double sigma = 2;


__global__ void calculate_x(double * in, double * out, int n, double step_t, double step_xyz, double sigma, int Nxyz)
{

	
	double * a = (double *)malloc(Nxyz * sizeof(double));
	double * b = (double *)malloc(Nxyz * sizeof(double));
	a[0] = 0;
	b[0] = sin((n * step_t) / 8);
		
	for (int j = 1; j < Nxyz - 1; j++) {
		a[j] = -(-sigma * step_t / step_xyz) / ((1 + 2 * sigma * step_t / step_xyz) + (-sigma * step_t / step_xyz) * a[j - 1]);
		b[j] = (in[threadIdx.y + threadIdx.y * threadIdx.x + threadIdx.y * threadIdx.x * (j - 1)] - (-sigma * step_t / step_xyz) * b[j - 1]) / ((1 + 2 * sigma * step_t / step_xyz) + (-sigma * step_t / step_xyz) * a[j - 1]);
	}
	out[threadIdx.y + threadIdx.y * threadIdx.x + threadIdx.y * threadIdx.x * (Nxyz - 1)] = sin((n * step_t) / 4);
	for (int j = Nxyz - 2; j > 0; j--) {
		out[threadIdx.y + threadIdx.y * threadIdx.x + threadIdx.y * threadIdx.x * j] = a[j] * out[threadIdx.y + threadIdx.y * threadIdx.x + threadIdx.y * threadIdx.x * (j + 1)] + b[j];
	}
	free(a);
	free(b);
	__syncthreads();
}

__global__ void calculate_y(double * in, double * out, int n, double step_t, double step_xyz, double sigma, int Nxyz)
{


	double * a = (double *)malloc(Nxyz * sizeof(double));
	double * b = (double *)malloc(Nxyz * sizeof(double));
	a[0] = 0;
	b[0] = sin((n * step_t) / 8);

	for (int k = 1; k < Nxyz - 1; k++) {
		a[k] = -(-sigma * step_t / step_xyz) / ((1 + 2 * sigma * step_t / step_xyz) + (-sigma * step_t / step_xyz) * a[k - 1]);
		b[k] = (in[threadIdx.y + threadIdx.y * (k - 1) + threadIdx.y * (k - 1) * threadIdx.x] - (-sigma * step_t / step_xyz) * b[k - 1]) / ((1 + 2 * sigma * step_t / step_xyz) + (-sigma * step_t / step_xyz) * a[k - 1]);
	}
	out[threadIdx.y + threadIdx.y * (Nxyz - 1) + threadIdx.y  * (Nxyz - 1) * threadIdx.x] = sin((n * step_t) / 4);
	for (int k = Nxyz - 2; k > 0; k--) {
		out[threadIdx.y + threadIdx.y * k + threadIdx.y * k * threadIdx.x ] = a[k] * out[threadIdx.y + threadIdx.y * (k + 1) + threadIdx.y * (k + 1) * threadIdx.x] + b[k];
	}
	free(a);
	free(b);
	__syncthreads();
}

__global__ void calculate_z(double * in, double * out, int n, double step_t, double step_xyz, double sigma, int Nxyz)
{


	double * a = (double *)malloc(Nxyz * sizeof(double));
	double * b = (double *)malloc(Nxyz * sizeof(double));
	a[0] = 0;
	b[0] = sin((n * step_t) / 8);

	for (int m = 1; m < Nxyz - 1; m++) {
		a[m] = -(-sigma * step_t / step_xyz) / ((1 + 2 * sigma * step_t / step_xyz) + (-sigma * step_t / step_xyz) * a[m - 1]);
		b[m] = (in[(m - 1) + (m - 1) * threadIdx.y + (m - 1) * threadIdx.y * threadIdx.x ] - (-sigma * step_t / step_xyz) * b[m - 1]) / ((1 + 2 * sigma * step_t / step_xyz) + (-sigma * step_t / step_xyz) * a[m - 1]);
	}
	out[(Nxyz - 1) + (Nxyz - 1) * threadIdx.y + (Nxyz - 1) * threadIdx.y * threadIdx.x] = sin((n * step_t) / 4);
	for (int m = Nxyz - 2; m > 0; m--) {
		out[m + m * threadIdx.y + m * threadIdx.y * threadIdx.x] = a[m] * out[(m + 1) + (m + 1) * threadIdx.y + (m + 1) * threadIdx.y * threadIdx.x] + b[m];
	}
	free(a);
	free(b);
	__syncthreads();
}

int main()
{
	setlocale(LC_ALL, "Russian");
	cout << "hello world!" << endl;
	double U[rasmer * Nt + 1][Nxyz][Nxyz][Nxyz];

	
	for (int j = 0; j < Nxyz; j++) {
		for (int k = 0; k < Nxyz; k++) {
			for (int m = 0; m < Nxyz; m++) {
				U[0][j][k][m] = 0;
			}
		}
	}
    
    double *row_in = new double [(Nxyz-2) * (Nxyz-2) * (Nxyz-2)];
	double *row_out = new double [(Nxyz-2) * (Nxyz-2) * (Nxyz-2)];

	double *row_in_gpu, *row_out_gpu;

	hipMalloc((void **)&row_in_gpu, sizeof(double) * (Nxyz-2) * (Nxyz-2) * (Nxyz-2));
	hipMalloc((void **)&row_out_gpu, sizeof(double) * (Nxyz-2) * (Nxyz-2) * (Nxyz-2));
	
	for (int n = 1; n <= rasmer; n ++) {
        for (int j = 1; j < Nxyz - 1; j++) {
			for (int k = 1; k < Nxyz - 1; k++) {
                for (int m = 1; m < Nxyz - 1; m++ ) {
					row_in[(m - 1) + (m - 1)*(k - 1) + (m - 1)*(k - 1)*(j - 1)] = U[n - 1][j][k][m];
                }
            }
        }

		hipMemcpy(row_in_gpu, row_in, sizeof(double) * (Nxyz - 2) * (Nxyz - 2) * (Nxyz - 2), hipMemcpyHostToDevice);

		dim3 gridDim(1);
		dim3 blockDim(Nxyz - 2, Nxyz - 2);

		calculate_x < << gridDim, blockDim >> >(row_in_gpu, row_out_gpu, n, step_t, step_xyz, sigma, Nxyz);

		hipMemcpy(row_out, row_out_gpu, sizeof(double) * (Nxyz - 2) * (Nxyz - 2) * (Nxyz - 2), hipMemcpyDeviceToHost);

		//посчитали x
        
		hipMemcpy(row_in_gpu, row_out, sizeof(double) * (Nxyz - 2) * (Nxyz - 2) * (Nxyz - 2), hipMemcpyHostToDevice);


		calculate_y < << gridDim, blockDim >> >(row_in_gpu, row_out_gpu, n, step_t, step_xyz, sigma, Nxyz);

		hipMemcpy(row_out, row_out_gpu, sizeof(double) * (Nxyz - 2) * (Nxyz - 2) * (Nxyz - 2), hipMemcpyDeviceToHost);
		
		//посчитали y

		hipMemcpy(row_in_gpu, row_out, sizeof(double) * (Nxyz - 2) * (Nxyz - 2) * (Nxyz - 2), hipMemcpyHostToDevice);


		calculate_z < << gridDim, blockDim >> >(row_in_gpu, row_out_gpu, n, step_t, step_xyz, sigma, Nxyz);

		hipMemcpy(row_out, row_out_gpu, sizeof(double) * (Nxyz - 2) * (Nxyz - 2) * (Nxyz - 2), hipMemcpyDeviceToHost);

		//посчитали z

		for (int j = 1; j < Nxyz - 1; j++) {
			for (int k = 1; k < Nxyz - 1; k++) {
				for (int m = 1; m < Nxyz - 1; m++) {
					U[n][j][k][m] = row_out[(m - 1) + (m - 1)*(k - 1) + (m - 1)*(k - 1)*(j - 1)];
				}
			}
		}
		cout << U[n][Nxyz - 2][Nxyz - 2][1];
	}
	
	_getch();
	return 0;
}