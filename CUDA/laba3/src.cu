#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <ctime>
#include <fstream>

using namespace std;

#define RASMER 2

__host__ int** sozdanie(int n, int m)
{
    int **A;
    A = new int *[n]; 
    for (int i = 0; i < n; i++) {
       A[i] = new int [m];
    }
    return A;
}

__host__ void initialize(int** A, int n, int m, int n_lim, int m_lim)
{
    srand(time(0));
    for(int i = 0; i < n_lim; i++ )
	{
	    for(int j = 0;  j < m_lim;  j++ )
		{
            if(i >= n || j >= m) {
                A[i][j] = 0;
            }
            else {
                A[i][j] = rand()%11 - 2 ;
            	cout.width(3);
            	cout << A[i][j] << "     ";
	    }
		}
    cout << endl;
	}
}

__global__ void block_proiz(int* A, int* B, int* C, int N, int M, int K) 
{
    int bx = blockIdx.x, by = blockIdx.y; 
    int tx = threadIdx.x, ty = threadIdx.y;
    int aBegin = N * RASMER * by;
    int aEnd = aBegin + N - 1;
    int bBegin = RASMER * bx;
    int aStep = RASMER, bStep = RASMER * K;
    int sum = 0;
    
    for ( int ia = aBegin, ib = bBegin; ia <= aEnd; ia += aStep, ib += bStep )
    {
        __shared__ float as [RASMER][RASMER];
        __shared__ float bs [RASMER][RASMER];
        as [ty][tx] = A [ia + N * ty + tx];
        bs [ty][tx] = B [ib + K * ty + tx]; 
        
        __syncthreads ();
        
        for ( int k = 0; k < RASMER; k++ ){
            sum += as [ty][k] * bs [k][tx]; 
        }
        __syncthreads ();
    }
    C [N * RASMER * by + RASMER * bx + K * ty + tx] = sum; 
}

int main()
{
    setlocale(LC_ALL, "Russian");
	int Ax(3), Ay(4), Bx(4), By(3);
	int N = Ax / RASMER;
	if (Ax % RASMER > 0) N++;

	int M = Ay / RASMER;
	if (Ay % RASMER > 0) M++;

	int K = By / RASMER;
	if (By % RASMER > 0) K++;

	int **A = sozdanie(RASMER * N, RASMER * M);
	initialize(A, Ax, Ay, RASMER * N, RASMER * M);
    
	cout << endl;
    
	int **B = sozdanie(RASMER * M, RASMER * K);
	initialize(B, Bx, By, RASMER * M, RASMER * K );

	int **C = sozdanie(RASMER * N, RASMER * K);

	cout << endl << "Matrix C: " << endl << endl;
    
    int size_A = (RASMER * N) * (RASMER * M);
	int size_B = (RASMER * M) * (RASMER * K);
	int size_C = (RASMER * N) * (RASMER * K);
    
    int* host_A = new int [size_A];
	int* host_B = new int [size_B];
	int* host_C = new int [size_C];
    
    for (int i = 0; i < RASMER * N; i++){
		for (int j = 0; j < RASMER * M; j++){
			host_A [i * RASMER * M + j] = A[i][j];
			cout << A[i][j] << " ";
		}
	}
	cout << endl;
	
	for (int i = 0; i < RASMER * M; i++){
		for (int j = 0; j < RASMER * K; j++){
			host_B [i * RASMERr * K + j] = B[i][j];
			cout << B[i][j] << " ";
		}
	}
	cout << endl;
    
    int *gpu_A, *gpu_B, *gpu_C;

    hipMalloc((void **)&gpu_A,  sizeof(int) * size_A); 
	hipMalloc((void **)&gpu_B,  sizeof(int) * size_B);
	hipMalloc((void **)&gpu_C,  sizeof(int) * size_C);

    hipMemcpy(gpu_A, host_A, sizeof(int) * size_A, hipMemcpyHostToDevice);
	hipMemcpy(gpu_B, host_B, sizeof(int) * size_B, hipMemcpyHostToDevice);

    dim3 gridDim(N, K); 
	dim3 blockDim(RASMER, RASMER);
    
    block_proiz <<< gridDim, blockDim >>> (gpu_A, gpu_B, gpu_C, N, M, K);
	
    hipMemcpy(host_C, gpu_C, N*sizeof(int), hipMemcpyDeviceToHost);	
    
    hipFree (gpu_A);
    hipFree (gpu_B);
    hipFree (gpu_C);

	for(int i = 0; i < Ax_number*By_number*N*K; i++) {
		cout << host_C[i] << " ";
	}
	return 0;
}

